
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAXN 1000

__device__ inline long long kpow(long long a,long long k)
{
	long long b=1;
	while(k)
	{
		if(k&1LL)b*=a;
		a*=a;
		k>>=1;
	}
	return b;
}

__global__ void add(long long *a,long long *b,long long *c)
{
	c[blockIdx.x]=kpow(a[blockIdx.x],b[blockIdx.x]);
	a[blockIdx.x]=threadIdx.x;
	b[blockIdx.x]=threadIdx.y;
}

long long a[MAXN],b[MAXN];

int main()
{
	for(int i=0;i<MAXN;i++)
	{
		a[i]=i;
		b[i]=4;
	}
	b[2]=10;
	b[3]=9;
	b[4]=5;
	b[8]=3;
	b[44]=3;
	long long *dev_a;
	long long *dev_b;
	long long *dev_ans;
	size_t siz=sizeof(long long)*MAXN;
	hipMalloc(&dev_a,siz);
	hipMalloc(&dev_b,siz);
	hipMalloc(&dev_ans,siz);
	hipMemcpy(dev_a,a,siz,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,siz,hipMemcpyHostToDevice);
	add<<<MAXN,1>>>(dev_a,dev_b,dev_ans);
	hipMemcpy(a,dev_ans,siz,hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_ans);
	for(int i=0;i<MAXN;i++)printf("%lld ",a[i]);
	printf("\n");
	return 0;
}
